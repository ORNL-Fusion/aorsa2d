#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"

#include "ccmplx.h"
#include "zcmplx.h"

#ifndef M_PI
#define M_PI 3.14159265358979323846264338327
#endif

#define INT int

//These are the types used for computation on the GPU
#define REAL float
#define CMPLX ccmplx

//These are the types used for referencing host memory
#define REAL_H double
#define CMPLX_H zcmplx

//This includes must follow the type declaration. I apologize, but this is the easiest way
#include "qlsum_gpu_kernels.cuh"

#define NUMBLOCKS 60

//Dimension constants
INT nkx1, nkx2,
  nky1, nky2,
  nzeta,
  nuper, nupar,
  nkdim1, nkdim2,
  mkdim1, mkdim2,
  lmaxdim;

//Device pointers
CMPLX
//Accumulator temporaries
*sum_p,
  *sum2_p, *sumkx2_p, *sumky2_p,
  *sumwdot_p, *sumwdotkx_p, *sumwdotky_p,
  //Various other temporaries
  *eps_t_p, *sumb_11_nm_t_p, *sumb_31_nm_t_p,
  *zbeta_p, *zbeta_iharm_p,
  //Output
  *sum_wdot_p, *sum_fx0_p, *sum_fy0_p,
  *b_sum_p, *c_sum_p, *e_sum_p, *f_sum_p,
  //Read only
  *xx_p, *yy_p,
  *ealphak_p, *ebetak_p, *ebk_p;

//All these pointers are read only
REAL *uper_p, *upar_p,
  *xkxsav_p, *xkysav_p,
  *dfduper_p, *dfdupar_p,
  *npara_sav_p,
  *xkperpn_tmp_p, *zetai_p, *Jni_p,
  *zetamin_tmp_p, *dzetai_tmp_p;

INT *nres_p, *mres_p;

//Single values
REAL sqmut0, dui, wcw;

//Various useful indexing constants
#define F 0
#define E 2
#define C 4
#define B 6

#define BUFF_SIZE 2048

float float_tmp[BUFF_SIZE];

void cudaMemcpyD2SH2D(void *targ_, void *source_, unsigned int n) {
  int i = 0, j = 0, blocks = n / BUFF_SIZE, remainder = n % BUFF_SIZE;
  float *targ;
  double *source;

  targ = (float *)targ_;
  source = (double *)source_;

  for(i = 0; i < blocks; i++) {
    for(j = 0; j < BUFF_SIZE; j++) {
      float_tmp[j] = (float)source[i * BUFF_SIZE + j];
    }

    hipMemcpy(&targ[i * BUFF_SIZE], &float_tmp[0], sizeof(float) * BUFF_SIZE, hipMemcpyHostToDevice);
  }

  if(remainder > 0) {
    for(j = 0; j < remainder; j++) {
      float_tmp[j] = (float)source[blocks * BUFF_SIZE + j];
    }

    hipMemcpy(&targ[blocks * BUFF_SIZE], &float_tmp[0], sizeof(float) * remainder, hipMemcpyHostToDevice);
  }
}

void cudaMemcpyS2DD2H(void *targ_, void *source_, unsigned int n) {
  int i = 0, j = 0, blocks = n / BUFF_SIZE, remainder = n % BUFF_SIZE;
  double *targ;
  float *source;

  targ = (double *)targ_;
  source = (float *)source_;

  for(i = 0; i < blocks; i++) {
    hipMemcpy(&float_tmp[0], &source[i * BUFF_SIZE], sizeof(float) * BUFF_SIZE, hipMemcpyDeviceToHost);

    for(j = 0; j < BUFF_SIZE; j++) {
      targ[i * BUFF_SIZE + j] = (double)float_tmp[j];
    }
  }

  if(remainder > 0) {
    hipMemcpy(&float_tmp[0], &source[i * BUFF_SIZE], sizeof(float) * remainder, hipMemcpyDeviceToHost);

    for(j = 0; j < remainder; j++) {
      targ[blocks * BUFF_SIZE + j] = (double)float_tmp[j];
    }
  }
}

//Description:
//Allocates all necessary GPU-side arrays and sets global dimension constants
extern "C" void qlsum_gpu_initialize_(INT *nuper_, INT *nupar_, INT *nzeta_,
				      INT *nkdim1_, INT *nkdim2_, INT *mkdim1_, INT *mkdim2_,
				      INT *lmaxdim_, INT *nkx1_, INT *nkx2_, INT *nky1_, INT *nky2_,
				      REAL_H *xkxsav_, REAL_H *xkysav_,
				      CMPLX_H *ealphak_, CMPLX_H *ebetak_, CMPLX_H *ebk_) {
  nkx1 = *nkx1_; nkx2 = *nkx2_;
  nky1 = *nky1_; nky2 = *nky2_;
  nzeta = *nzeta_;
  nuper = *nuper_; nupar = *nupar_;
  nkdim1 = *nkdim1_; nkdim2 = *nkdim2_;
  mkdim1 = *mkdim1_; mkdim2 = *mkdim2_;
  lmaxdim = *lmaxdim_;

  //Exercise the hipMalloc function
  hipMalloc((void **)&sum_p, 8 * sizeof(CMPLX) * nuper * nupar * NUMBLOCKS);

  hipMalloc((void **)&xx_p, sizeof(CMPLX) * (nkdim2 - nkdim1 + 1));
  hipMalloc((void **)&yy_p, sizeof(CMPLX) * (mkdim2 - mkdim1 + 1));
  hipMalloc((void **)&ealphak_p, sizeof(CMPLX) * (nkdim2 - nkdim1 + 1) * (mkdim2 - mkdim1 + 1));
  hipMalloc((void **)&ebetak_p, sizeof(CMPLX) * (nkdim2 - nkdim1 + 1) * (mkdim2 - mkdim1 + 1));
  hipMalloc((void **)&ebk_p, sizeof(CMPLX) * (nkdim2 - nkdim1 + 1) * (mkdim2 - mkdim1 + 1));

  hipMalloc((void **)&sum_wdot_p, sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMalloc((void **)&sum_fx0_p, sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMalloc((void **)&sum_fy0_p, sizeof(CMPLX) * nuper * NUMBLOCKS);

  hipMalloc((void **)&b_sum_p, sizeof(CMPLX) * nuper * nupar * NUMBLOCKS);
  hipMalloc((void **)&c_sum_p, sizeof(CMPLX) * nuper * nupar * NUMBLOCKS);
  hipMalloc((void **)&e_sum_p, sizeof(CMPLX) * nuper * nupar * NUMBLOCKS);
  hipMalloc((void **)&f_sum_p, sizeof(CMPLX) * nuper * nupar * NUMBLOCKS);

  hipMalloc((void **)&nres_p, sizeof(INT) * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1));
  hipMalloc((void **)&mres_p, sizeof(INT) * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1));
  hipMalloc((void **)&zbeta_p, sizeof(CMPLX) * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1));
  hipMalloc((void **)&zbeta_iharm_p, sizeof(CMPLX) * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1));

  hipMalloc((void **)&uper_p, sizeof(REAL) * nuper);
  hipMalloc((void **)&upar_p, sizeof(REAL) * nupar);
  hipMalloc((void **)&xkxsav_p, sizeof(REAL) * (nkdim2 - nkdim1 + 1));
  hipMalloc((void **)&xkysav_p, sizeof(REAL) * (mkdim2 - mkdim1 + 1));
  hipMalloc((void **)&dfduper_p, sizeof(REAL) * (nuper * nupar));
  hipMalloc((void **)&dfdupar_p, sizeof(REAL) * (nuper * nupar));
  hipMalloc((void **)&npara_sav_p, sizeof(REAL) * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1));

  hipMalloc((void **)&sum2_p, 3 * sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMalloc((void **)&sumkx2_p, 3 * sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMalloc((void **)&sumky2_p, 3 * sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMalloc((void **)&sumwdot_p, 2 * sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMalloc((void **)&sumwdotkx_p, 2 * sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMalloc((void **)&sumwdotky_p, 2 * sizeof(CMPLX) * nuper * NUMBLOCKS);

  hipMalloc((void **)&xkperpn_tmp_p, sizeof(REAL) * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1));
  hipMalloc((void **)&zetai_p, sizeof(REAL) * (nzeta + 1));
  hipMalloc((void **)&Jni_p, sizeof(REAL) * nuper * (2 * lmaxdim + 1) * (nzeta + 1));
  hipMalloc((void **)&zetamin_tmp_p, sizeof(REAL) * nuper);
  hipMalloc((void **)&dzetai_tmp_p, sizeof(REAL) * nuper);

  hipMalloc((void **)&eps_t_p, 3 * sizeof(CMPLX) * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1));

  hipMalloc((void **)&sumb_11_nm_t_p, sizeof(CMPLX) * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1) * nuper);
  hipMalloc((void **)&sumb_31_nm_t_p, sizeof(CMPLX) * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1) * nuper);

  //Copy over useful arrays
  cudaMemcpyD2SH2D(ealphak_p, ealphak_, 2 * (nkdim2 - nkdim1 + 1) * (mkdim2 - mkdim1 + 1));
  cudaMemcpyD2SH2D(ebetak_p, ebetak_, 2 * (nkdim2 - nkdim1 + 1) * (mkdim2 - mkdim1 + 1));
  cudaMemcpyD2SH2D(ebk_p, ebk_, 2 * (nkdim2 - nkdim1 + 1) * (mkdim2 - mkdim1 + 1));

  cudaMemcpyD2SH2D(xkxsav_p, xkxsav_, (nkdim2 - nkdim1 + 1));
  cudaMemcpyD2SH2D(xkysav_p, xkysav_, (mkdim2 - mkdim1 + 1));
}

//Description:
//Frees all dynamic arrays
extern "C" void qlsum_gpu_cleanup_() {
  hipFree(sum_p);

  hipFree(xx_p);
  hipFree(yy_p);
  hipFree(ealphak_p);
  hipFree(ebetak_p);
  hipFree(ebk_p);

  hipFree(sum_wdot_p);
  hipFree(sum_fx0_p);
  hipFree(sum_fy0_p);

  hipFree(b_sum_p);
  hipFree(c_sum_p);
  hipFree(e_sum_p);
  hipFree(f_sum_p);

  hipFree(nres_p);
  hipFree(mres_p);
  hipFree(zbeta_p);
  hipFree(zbeta_iharm_p);

  hipFree(uper_p);
  hipFree(upar_p);
  hipFree(xkxsav_p);
  hipFree(xkysav_p);
  hipFree(dfduper_p);
  hipFree(dfdupar_p);
  hipFree(npara_sav_p);

  hipFree(sum2_p);
  hipFree(sumkx2_p);
  hipFree(sumky2_p);
  hipFree(sumwdot_p);
  hipFree(sumwdotkx_p);
  hipFree(sumwdotky_p);

  hipFree(xkperpn_tmp_p);
  hipFree(zetai_p);
  hipFree(Jni_p);
  hipFree(zetamin_tmp_p);
  hipFree(dzetai_tmp_p);

  hipFree(eps_t_p);

  hipFree(sumb_11_nm_t_p);
  hipFree(sumb_31_nm_t_p);
}	

/*
  Description:
  Computes the shared epsx/epsy/epsz values

  There are iresmax epsx/epsy/epsz values generated
  To save pointer space, they are saved in one large array (eps_t)

  Because iresmax is at most (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1), we allocate
  an array of 3 * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1) elements, and say that
  the first (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1) are for epsx values,
  the second (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1) are for epsy values, etc.

  Output:
  eps_t

  Parallelism:
  There is one loop which operates across the resonant points

  Each block of threads takes a chunk of the 0 -> iresmax - 1 loop
  For each of these chunks, each thread in a block takes a consecutive
  element to operate on. There is no need for the chunksize to be a multiple
  of the threadsize.
*/
__global__ void qlsum_gpu_iharm_shared(INT nkx1, INT nkx2,
				       INT nky1, INT nky2,
				       INT nkdim1, INT nkdim2,
				       INT mkdim1,
				       INT iresmax, INT *nres, INT *mres,
				       CMPLX *zbeta, CMPLX *zbeta_iharm,
				       CMPLX *xx, CMPLX *yy,
				       CMPLX *ealphak, CMPLX *ebetak, CMPLX *ebk,
				       CMPLX *eps_t) {
  INT k_uper = threadIdx.x, block = blockIdx.x,
    ires, iresstart, iresfinish;

  INT xy_ind, dim_ind,
    n, m;

  CMPLX epsx, epsy, epsz,
    cexp1, cexp2, cexp0;

  iresstart = min((iresmax + NUMBLOCKS - 1) / NUMBLOCKS * block, iresmax);
  iresfinish = min(((iresmax + NUMBLOCKS - 1) / NUMBLOCKS) * (block + 1), iresmax);

  for(ires = iresstart + k_uper; ires < iresfinish; ires += blockDim.x) {
    n = nres[ires];
    m = mres[ires];

    xy_ind = (n - nkx1) + (m - nky1) * (nkx2 - nkx1 + 1);
    dim_ind = (n - nkdim1) + (m - mkdim1) * (nkdim2 - nkdim1 + 1);

    cexp2 = xx[(n - nkx1)] * yy[(m - nky1)] * zbeta_iharm[xy_ind];
    cexp0 = cexp2 * zbeta[xy_ind];
    cexp1 = cexp0 * zbeta[xy_ind];
	
    epsx = (ealphak[dim_ind] - zcmplx(0.0, 1.0) * ebetak[dim_ind]) * cexp1 / sqrtf(2.0f);
    epsy = (ealphak[dim_ind] + zcmplx(0.0, 1.0) * ebetak[dim_ind]) * cexp2 / sqrtf(2.0f);
    epsz = ebk[dim_ind] * cexp0;
      
    eps_t[ires + 0 * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1)] = epsx;
    eps_t[ires + 1 * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1)] = epsy;
    eps_t[ires + 2 * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1)] = epsz;
  }
}

/*
  Description:
  Computes various sums used in final reduction/generation of output
  Stores temporary copy of values derived from Bessel function interpolation
  (used by qlsum_gpu_iharm_second)

  Multiple arrays are stored within one array pointer to save argument space
  sum2 expands to sum2_1, sum2_2, and sum2_3 in the Fortran code
  sumkx2 and sumky2 are similar
  sum2_2 is accessed nuper * NUMBLOCKS elements from &sum[0]

  Output:
  sumb_11_nm_t, sumb_31_nm_t
  sum2, sumkx2, sumky2

  Parallelism:
  There is one explicit loop which operates across the resonant points

  Each block takes a chunk of this loop, which they work through in a +1 pattern
  Each thread within a block is assigned a k_uper value from 0 to nuper - 1
  Effectively, a nuper wide SIMD unit operates across the resonant points

  Because values must be accumulated across the resonant points, there must
  be a reduction across each of the sum2, sumkx2, and sumky2 data
*/
__global__ void qlsum_gpu_iharm_first(INT nkx1, INT nkx2,
				      INT nky1, INT nky2,
				      INT nkdim1, INT mkdim1,
				      INT nuper, INT nupar,
				      INT iharm, INT iresmax,
				      INT lmaxdim,
				      REAL nwcw,
				      REAL sqmut0, REAL dui, CMPLX *eps_t,
				      CMPLX *sumb_11_nm_t, CMPLX *sumb_31_nm_t,
				      INT *nres, INT *mres,
				      CMPLX *sum2, CMPLX *sumkx2, CMPLX *sumky2,
				      REAL *uper, REAL *upar,
				      REAL *xkxsav, REAL *xkysav,
				      REAL *dfduper, REAL *npara_sav,
				      REAL *xkperpn_tmp, REAL *zetai, REAL *Jni,
				      REAL *zetamin_tmp, REAL *dzetai_tmp) {
  INT k_uper = threadIdx.x, block = blockIdx.x,
    ires = 0, i2,
    idx2, idx3,
    iresstart, iresfinish;
  __shared__ INT xy_ind, n, m;

  REAL uper_kuper, zeta0, p2, zetamin, dzetai,
    Jnp_t, Jnm_t, Jnn_t;
  __shared__ REAL upar0, factor,  xkxsav_n, xkysav_m;

  CMPLX sum2_1 = 0.0, sum2_2 = 0.0, sum2_3 = 0.0,
    sumkx2_1 = 0.0, sumkx2_2, sumkx2_3 = 0.0,
    sumky2_1 = 0.0, sumky2_2, sumky2_3 = 0.0,
    sumb_11_nm = 0.0, sumb_31_nm = 0.0,
    sumwdot_11 = 0.0, sumwdot_31 = 0.0,
    sumwdotkx_11 = 0.0, sumwdotkx_31 = 0.0,
    sumwdotky_11 = 0.0, sumwdotky_31 = 0.0;
  __shared__ CMPLX epsx, epsy, epsz;

  __syncthreads(); //Every thread calls the __shared__ CMPLX constructor

  iresstart = min((iresmax + NUMBLOCKS - 1) / NUMBLOCKS * block, iresmax);
  iresfinish = min(((iresmax + NUMBLOCKS - 1) / NUMBLOCKS) * (block + 1), iresmax);

  uper_kuper = uper[k_uper];
  zetamin = zetamin_tmp[k_uper];
  dzetai = dzetai_tmp[k_uper];

  for(ires = iresstart; ires < iresfinish; ires++) {
    if(k_uper == 0) {
      n = nres[ires];
      m = mres[ires];

      xy_ind = (n - nkx1) + (m - nky1) * (nkx2 - nkx1 + 1);

      epsx = eps_t[ires + 0 * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1)];
      epsy = eps_t[ires + 1 * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1)];
      epsz = eps_t[ires + 2 * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1)];

      upar0 = sqmut0 / npara_sav[xy_ind] * (1.0 - nwcw);
      factor = M_PI * sqmut0 / abs(npara_sav[xy_ind]);

      xkxsav_n = xkxsav[n - nkdim1];
      xkysav_m = xkysav[m - mkdim1];
    }
    __syncthreads();

    zeta0 = xkperpn_tmp[(n - nkx1) + (m - nky1) * (nkx2 - nkx1 + 1)] * uper_kuper;
  
    i2 = int((zeta0 - zetamin) * dzetai);
    p2 = (zeta0 - (zetamin + (REAL)i2 / dzetai)) * dzetai;

    idx2 = k_uper + (iharm + lmaxdim) * nuper + i2 * (2 * lmaxdim + 1) * nuper;
    idx3 = k_uper + (iharm + lmaxdim) * nuper + (i2 + 1) * (2 * lmaxdim + 1) * nuper;
      
    Jnm_t = Jni[idx2 - nuper] + p2 * (Jni[idx3 - nuper] - Jni[idx2 - nuper]);
    Jnn_t = Jni[idx2] + p2 * (Jni[idx3] - Jni[idx2]);
    Jnp_t = Jni[idx2 + nuper] + p2 * (Jni[idx3 + nuper] - Jni[idx2 + nuper]);

    sum2_1 += ~epsx * Jnp_t; 
    sum2_2 += ~epsy * Jnm_t;
    sum2_3 += ~epsz * Jnn_t;

    sumkx2_1 += ~epsx * Jnp_t * xkxsav_n;
    sumkx2_2 += ~epsy * Jnm_t * xkxsav_n;
    sumkx2_3 += ~epsz * Jnn_t * xkxsav_n;

    sumky2_1 += ~epsx * Jnp_t * xkysav_m;
    sumky2_2 += ~epsy * Jnm_t * xkysav_m;
    sumky2_3 += ~epsz * Jnn_t * xkysav_m;

    sumb_11_nm = (epsx * uper_kuper * uper_kuper * Jnp_t + epsy * uper_kuper * uper_kuper  * Jnm_t + epsz * sqrtf(2.0f) * uper_kuper * upar0 * Jnn_t) * factor;
    sumb_31_nm = (epsx * sqrtf(2.0f) * uper_kuper * upar0 * Jnp_t + epsy * sqrtf(2.0f) * uper_kuper * upar0 * Jnm_t + epsz * 2.0 * upar0 * upar0 * Jnn_t) * factor;

    sumb_11_nm_t[k_uper + ires * nuper] = sumb_11_nm;
    sumb_31_nm_t[k_uper + ires * nuper] = sumb_31_nm;

    __syncthreads();
  }

  sum2[k_uper + block * nuper + 0 * nuper * NUMBLOCKS] += sum2_1; sum2[k_uper + block * nuper + 1 * nuper * NUMBLOCKS] += sum2_2; sum2[k_uper + block * nuper + 2 * nuper * NUMBLOCKS] += sum2_3;
  sumkx2[k_uper + block * nuper + 0 * nuper * NUMBLOCKS] += sumkx2_1; sumkx2[k_uper + block * nuper + 1 * nuper * NUMBLOCKS] += sumkx2_2; sumkx2[k_uper + block * nuper + 2 * nuper * NUMBLOCKS] += sumkx2_3;
  sumky2[k_uper + block * nuper + 0 * nuper * NUMBLOCKS] += sumky2_1; sumky2[k_uper + block * nuper + 1 * nuper * NUMBLOCKS] += sumky2_2; sumky2[k_uper + block * nuper + 2 * nuper * NUMBLOCKS] += sumky2_3;
}

/*
  Description:
  Computes various sums used in final reduction/generation of output

  Multiple arrays are stored within one array pointer to save argument space
  sum expands to sumb_11, sumb_31, sumc_11, sumc_31, sume_11, sume_31,
  sumf_11, and sumf_31 in the Fortran code
  The '31' elements are offset from the '11' elements by nuper * nupar * NUMBLOCKS
  The letters are offset from the base pointer by [B, C, E, or F] * nuper * nupar * NUMBLOCKS

  sumwdot expands to sumwdot_11, and sumwdot_31 in the Fortran code
  sumwdotkx and sumwdotky are similar
  The '31' elements are offset from the '11' elements by nuper * NUMBLOCKS

  Output:
  sum, sumwdot, sumwdotkx, sumwdotky

  Parallelism:
  There is one explicit loop which operates across the resonant points

  Each block takes a chunk of this loop, which they work through in a +1 pattern
  Each thread within a block is assigned a k_uper value from 0 to nuper - 1
  Effectively, a nuper wide SIMD unit operates across the resonant points

  Because values must be accumulated across the resonant points, there must
  be a reduction across each of effective arrays in the the sum, sumwdot,
  sumwdotkx, and sumwdotky data
*/
__global__ void qlsum_gpu_iharm_second(INT nkx1, INT nkx2,
				       INT nky1, INT nky2,
				       INT nkdim1, INT mkdim1,
				       INT nuper, INT nupar,
				       INT iharm, INT iresmax,
				       REAL nwcw,
				       REAL sqmut0, REAL dui, CMPLX *eps_t,
				       CMPLX *sumb_11_nm_t, CMPLX *sumb_31_nm_t,
				       INT *nres, INT *mres,
				       CMPLX *sum,
				       CMPLX *sumwdot, CMPLX *sumwdotkx, CMPLX *sumwdotky,
				       REAL *uper, REAL *upar,
				       REAL *xkxsav, REAL *xkysav,
				       REAL *dfduper, REAL *dfdupar,
				       REAL *npara_sav,
				       REAL *xkperpn_tmp, REAL *zetai, REAL *Jni,
				       REAL *zetamin_tmp, REAL *dzetai_tmp) {

  INT k_uper = threadIdx.x, block = blockIdx.x, ires = 0, iresstart, iresfinish;
  __shared__ INT i, xy_ind, n, m;

  REAL u, sinth, facte, u0, uper_kuper,
    dfduper0, dfdupar0,
    dfactpar, dfactper;
  __shared__ REAL npara1, upar0, p, xkxsav_n, xkysav_m;

  CMPLX sumf_11_t = 0.0, sumf_31_t = 0.0,
    sume_11_t = 0.0, sume_31_t = 0.0,
    sumc_11_t = 0.0, sumc_31_t = 0.0,
    sumb_11_t = 0.0, sumb_31_t = 0.0,
    sumb_11_nm = 0.0, sumb_31_nm = 0.0,
    sumwdot_11 = 0.0, sumwdot_31 = 0.0,
    sumwdotkx_11 = 0.0, sumwdotkx_31 = 0.0,
    sumwdotky_11 = 0.0, sumwdotky_31 = 0.0;
  __shared__ CMPLX epsx, epsy, epsz;

  __syncthreads(); //Every thread calls the __shared__ CMPLX constructor

  iresstart = min((iresmax + NUMBLOCKS - 1) / NUMBLOCKS * block, iresmax);
  iresfinish = min(((iresmax + NUMBLOCKS - 1) / NUMBLOCKS) * (block + 1), iresmax);

  uper_kuper = uper[k_uper];

  if(k_uper == 0) {
    n = nres[0];
    m = mres[0];

    xy_ind = (n - nkx1) + (m - nky1) * (nkx2 - nkx1 + 1);

    upar0 = sqmut0 / npara_sav[xy_ind] * (1.0 - nwcw);
    i = (int)floor((upar0 - upar[0]) * dui);
  }

  __syncthreads();

  for(ires = iresstart; ires < iresfinish; ires++) {
    if(k_uper == 0) {
      n = nres[ires];
      m = mres[ires];

      xy_ind = (n - nkx1) + (m - nky1) * (nkx2 - nkx1 + 1);
      npara1 = npara_sav[xy_ind];

      epsx = eps_t[ires + 0 * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1)];
      epsy = eps_t[ires + 1 * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1)];
      epsz = eps_t[ires + 2 * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1)];

      upar0 = sqmut0 / npara1 * (1.0 - nwcw);
      p = (upar0 - upar[i]) * dui;

      xkxsav_n = xkxsav[n - nkdim1];
      xkysav_m = xkysav[m - mkdim1];
    }
    __syncthreads();

    u = sqrt(upar0 * upar0 + uper_kuper * uper_kuper) + 0.00000001;
    sinth = uper_kuper * __frcp_rn((float)u) + 0.00000001;      
    facte = (nwcw - sinth * sinth) * __frcp_rn((float)upar0);
    
    //// This is the non-Maxwellian version (which covers the Maxwellian case)
    // Replacing the u0 = ... statement with
    // u0 = dfduper0 gives the Maxwellian qlsum
    //
    if(i <= (nupar - 1)) {
      dfduper0 = dfduper[k_uper + i * nuper] + (dfduper[k_uper + (i + 1) * nuper] - dfduper[k_uper + i * nuper]) * p;
      dfdupar0 = dfdupar[k_uper + i * nuper] + (dfdupar[k_uper + (i + 1) * nuper] - dfdupar[k_uper + i * nuper]) * p;
    } else {
      dfduper0 = dfduper[k_uper + i * nuper];
      dfdupar0 = dfdupar[k_uper + i * nuper];
    }

    dfactpar = npara1 * upar0 * __frcp_rn((float)sqmut0);
    dfactper = npara1 * uper_kuper * __frcp_rn((float)sqmut0);
    
    u0 = (1.0 - dfactpar) * dfduper0 + dfactper * dfdupar0;
    ////

    REAL sinth_inv = __frcp_rn((float)sinth);

    sumb_11_nm = sumb_11_nm_t[k_uper + ires * nuper];
    sumb_11_t += sumb_11_nm;
    sume_11_t += sumb_11_nm * facte;
    sumc_11_t += sumb_11_nm * facte * sinth_inv;
    sumf_11_t += sumb_11_nm * facte * facte * sinth_inv;
    sumwdot_11 += sumb_11_nm * u0;
    sumwdotkx_11 += sumb_11_nm * xkxsav_n * u0;
    sumwdotky_11 += sumb_11_nm * xkysav_m * u0;

    sumb_31_nm = sumb_31_nm_t[k_uper + ires * nuper];
    sumb_31_t += sumb_31_nm;
    sume_31_t += sumb_31_nm * facte;
    sumc_31_t += sumb_31_nm * facte * sinth_inv;
    sumf_31_t += sumb_31_nm * facte * facte * sinth_inv;
    sumwdot_31 += sumb_31_nm * u0;
    sumwdotkx_31 += sumb_31_nm * xkxsav_n * u0;
    sumwdotky_31 += sumb_31_nm * xkysav_m * u0;

    __syncthreads();
  }

  sum[k_uper + i * nuper + block * nuper * nupar + (F + 0) * nuper * nupar * NUMBLOCKS] += sumf_11_t;
  sum[k_uper + i * nuper + block * nuper * nupar + (F + 1) * nuper * nupar * NUMBLOCKS] += sumf_31_t;

  sum[k_uper + i * nuper + block * nuper * nupar + (E + 0) * nuper * nupar * NUMBLOCKS] += sume_11_t;
  sum[k_uper + i * nuper + block * nuper * nupar + (E + 1) * nuper * nupar * NUMBLOCKS] += sume_31_t;

  sum[k_uper + i * nuper + block * nuper * nupar + (C + 0) * nuper * nupar * NUMBLOCKS] += sumc_11_t;
  sum[k_uper + i * nuper + block * nuper * nupar + (C + 1) * nuper * nupar * NUMBLOCKS] += sumc_31_t;

  sum[k_uper + i * nuper + block * nuper * nupar + (B + 0) * nuper * nupar * NUMBLOCKS] += sumb_11_t;
  sum[k_uper + i * nuper + block * nuper * nupar + (B + 1) * nuper * nupar * NUMBLOCKS] += sumb_31_t;

  sumwdot[k_uper + block * nuper + 0 * nuper * NUMBLOCKS] += sumwdot_11;
  sumwdot[k_uper + block * nuper + 1 * nuper * NUMBLOCKS] += sumwdot_31;

  sumwdotkx[k_uper + block * nuper + 0 * nuper * NUMBLOCKS] += sumwdotkx_11;
  sumwdotkx[k_uper + block * nuper + 1 * nuper * NUMBLOCKS] += sumwdotkx_31;

  sumwdotky[k_uper + block * nuper + 0 * nuper * NUMBLOCKS] += sumwdotky_11;
  sumwdotky[k_uper + block * nuper + 1 * nuper * NUMBLOCKS] += sumwdotky_31;
}

__device__ CMPLX reduceCMPLX(CMPLX *array, int stride, int n) {
  INT i = 0;
  CMPLX acc = 0.0;

  for(i = 0; i < n; i++) {
    acc += array[i * stride];
  }

  return acc;
}

/*
  Description:
  Performs the reduction and calculates the qlsum output values

  Output:
  sum_wdot, sum_fx0, sum_fy0, b_sum, c_sum, e_sum, f_sum

  Parallelism:
  The reduction operates as a single nuper wide SIMD unit which runs along,
  reduces variables as necessary, and produces output

  This code only uses one multiprocessor
*/
__global__ void qlsum_gpu_iharm_reduction(INT nuper, INT nupar,
					  CMPLX *sum_wdot, CMPLX *sum_fx0, CMPLX *sum_fy0,
					  CMPLX *b_sum, CMPLX *c_sum, CMPLX *e_sum, CMPLX *f_sum,
					  CMPLX *sum,
					  CMPLX *sum2, CMPLX *sumkx2, CMPLX *sumky2,
					  CMPLX *sumwdot, CMPLX *sumwdotkx, CMPLX *sumwdotky) {
  INT k_uper = threadIdx.x, i = 0, j = 0, i_uprl = 0;

  CMPLX sumwdot_11 = 0.0, sumwdot_31 = 0.0,
    sumwdotkx_11 = 0.0, sumwdotkx_31 = 0.0,
    sumwdotky_11 = 0.0, sumwdotky_31 = 0.0,
    sum2_1 = 0.0, sum2_2 = 0.0, sum2_3 = 0.0,
    sumkx2_1 = 0.0, sumkx2_2 = 0.0, sumkx2_3 = 0.0,
    sumky2_1 = 0.0, sumky2_2 = 0.0, sumky2_3 = 0.0;
  
  //Perform reduction
  sumwdot_11 = reduceCMPLX(&sumwdot[k_uper + 0 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);
  sumwdot_31 = reduceCMPLX(&sumwdot[k_uper + 1 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);

  sumwdotkx_11 = reduceCMPLX(&sumwdotkx[k_uper + 0 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);
  sumwdotky_31 = reduceCMPLX(&sumwdotkx[k_uper + 1 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);

  sumwdotky_11 = reduceCMPLX(&sumwdotky[k_uper + 0 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);
  sumwdotky_31 = reduceCMPLX(&sumwdotky[k_uper + 1 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);

  sum2_1 = reduceCMPLX(&sum2[k_uper + 0 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);
  sum2_2 = reduceCMPLX(&sum2[k_uper + 1 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);
  sum2_3 = reduceCMPLX(&sum2[k_uper + 2 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);

  sumkx2_1 = reduceCMPLX(&sumkx2[k_uper + 0 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);
  sumkx2_2 = reduceCMPLX(&sumkx2[k_uper + 1 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);
  sumkx2_3 = reduceCMPLX(&sumkx2[k_uper + 2 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);

  sumky2_1 = reduceCMPLX(&sumky2[k_uper + 0 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);
  sumky2_2 = reduceCMPLX(&sumky2[k_uper + 1 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);
  sumky2_3 = reduceCMPLX(&sumky2[k_uper + 2 * nuper * NUMBLOCKS], nuper, NUMBLOCKS);
  
  //Calculate output
  sum_wdot[k_uper] +=
    sum2_1 * sumwdot_11 +
    sum2_2 * sumwdot_11 +
    sum2_3 * sumwdot_31;

  sum_fx0[k_uper] +=
    sumkx2_1 * sumwdot_11 +
    sumkx2_2 * sumwdot_11 +
    sumkx2_3 * sumwdot_31 +
    sum2_1 * sumwdotkx_11 +
    sum2_2 * sumwdotkx_11 +
    sum2_3 * sumwdotkx_31;

  sum_fy0[k_uper] +=
    sumky2_1 * sumwdot_11 +
    sumky2_2 * sumwdot_11 +
    sumky2_3 * sumwdot_31 +
    sum2_1 * sumwdotky_11 +
    sum2_2 * sumwdotky_11 +
    sum2_3 * sumwdotky_31;

  //Perform reduction
  for(j = 0; j < nupar; j++) {
    sum[k_uper + j * nuper + (B + 0) * nuper * nupar * NUMBLOCKS] = reduceCMPLX(&sum[k_uper + j * nuper + (B + 0) * nuper * nupar * NUMBLOCKS], nupar * nuper, NUMBLOCKS);
    sum[k_uper + j * nuper + (B + 1) * nuper * nupar * NUMBLOCKS] = reduceCMPLX(&sum[k_uper + j * nuper + (B + 1) * nuper * nupar * NUMBLOCKS], nupar * nuper, NUMBLOCKS);
      
    sum[k_uper + j * nuper + (C + 0) * nuper * nupar * NUMBLOCKS] = reduceCMPLX(&sum[k_uper + j * nuper + (C + 0) * nuper * nupar * NUMBLOCKS], nupar * nuper, NUMBLOCKS);
    sum[k_uper + j * nuper + (C + 1) * nuper * nupar * NUMBLOCKS] = reduceCMPLX(&sum[k_uper + j * nuper + (C + 1) * nuper * nupar * NUMBLOCKS], nupar * nuper, NUMBLOCKS);
      
    sum[k_uper + j * nuper + (E + 0) * nuper * nupar * NUMBLOCKS] = reduceCMPLX(&sum[k_uper + j * nuper + (E + 0) * nuper * nupar * NUMBLOCKS], nupar * nuper, NUMBLOCKS);
    sum[k_uper + j * nuper + (E + 1) * nuper * nupar * NUMBLOCKS] = reduceCMPLX(&sum[k_uper + j * nuper + (E + 1) * nuper * nupar * NUMBLOCKS], nupar * nuper, NUMBLOCKS);
      
    sum[k_uper + j * nuper + (F + 0) * nuper * nupar * NUMBLOCKS] = reduceCMPLX(&sum[k_uper + j * nuper + (F + 0) * nuper * nupar * NUMBLOCKS], nupar * nuper, NUMBLOCKS);
    sum[k_uper + j * nuper + (F + 1) * nuper * nupar * NUMBLOCKS] = reduceCMPLX(&sum[k_uper + j * nuper + (F + 1) * nuper * nupar * NUMBLOCKS], nupar * nuper, NUMBLOCKS);
  }

  //Calculate output
  for(i_uprl = 0; i_uprl < nupar; i_uprl++) {
    b_sum[k_uper + i_uprl * nuper] +=
      sum2_1 * sum[k_uper + i_uprl * nuper + (B + 0) * nuper * nupar * NUMBLOCKS] +
      sum2_2 * sum[k_uper + i_uprl * nuper + (B + 0) * nuper * nupar * NUMBLOCKS] +
      sum2_3 * sum[k_uper + i_uprl * nuper + (B + 1) * nuper * nupar * NUMBLOCKS];

    c_sum[k_uper + i_uprl * nuper] +=
      sum2_1 * sum[k_uper + i_uprl * nuper + (C + 0) * nuper * nupar * NUMBLOCKS] +
      sum2_2 * sum[k_uper + i_uprl * nuper + (C + 0) * nuper * nupar * NUMBLOCKS] +
      sum2_3 * sum[k_uper + i_uprl * nuper + (C + 1) * nuper * nupar * NUMBLOCKS];

    e_sum[k_uper + i_uprl * nuper] +=
      sum2_1 * sum[k_uper + i_uprl * nuper + (E + 0) * nuper * nupar * NUMBLOCKS] +
      sum2_2 * sum[k_uper + i_uprl * nuper + (E + 0) * nuper * nupar * NUMBLOCKS] +
      sum2_3 * sum[k_uper + i_uprl * nuper + (E + 1) * nuper * nupar * NUMBLOCKS];

    f_sum[k_uper + i_uprl * nuper] +=
      sum2_1 * sum[k_uper + i_uprl * nuper + (F + 0) * nuper * nupar * NUMBLOCKS] +
      sum2_2 * sum[k_uper + i_uprl * nuper + (F + 0) * nuper * nupar * NUMBLOCKS] +
      sum2_3 * sum[k_uper + i_uprl * nuper + (F + 1) * nuper * nupar * NUMBLOCKS];

  }
}

/*
  Description:
  Updates the zbeta values by multiplying in respective elements of zbeta_iharm

  Output:
  zbeta

  Parallelism:
  There is one loop which operates across the grid points (zbeta and zbeta_iharm
  really point to 2D arrays)

  Each block of threads takes a chunk of the 0 -> n - 1 loop
  For each of these chunks, each thread in a block takes a consecutive
  element to operate on. There is no need for the chunksize to be a multiple
  of the threadsize.
*/
__global__ void qlsum_gpu_iharm_zbeta_update(INT n, CMPLX *zbeta, CMPLX *zbeta_iharm) {
  INT thread = threadIdx.x, block = blockIdx.x, numthreads = blockDim.x, start, finish, i;

  start = min((n + NUMBLOCKS - 1) / NUMBLOCKS * block, n);
  finish = min(((n + NUMBLOCKS - 1) / NUMBLOCKS) * (block + 1), n);

  for(i = start + thread; i < finish; i += numthreads) {
    zbeta_iharm[i] *= zbeta[i];
  }
}

//Description:
//Initializes all the reduction temporaries
void qlsum_cpu_iharm_setup() {
  hipMemset(sum_p, 0, 8 * sizeof(CMPLX) * nuper * nupar * NUMBLOCKS);
  hipMemset(sum2_p, 0, 3 * sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMemset(sumkx2_p, 0, 3 * sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMemset(sumky2_p, 0, 3 * sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMemset(sumwdot_p, 0, 2 * sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMemset(sumwdotkx_p, 0, 2 * sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMemset(sumwdotky_p, 0, 2 * sizeof(CMPLX) * nuper * NUMBLOCKS);
}

//Description:
//Provides a wrapper around the GPU grunt work kernel calls which process
// a worklist for a given iharm number
void qlsum_cpu_iharm_accumulate(INT iharm, INT iresmax,
				INT *nres, INT *mres) {
  dim3 threads(nuper);
  dim3 blocks(NUMBLOCKS);

  if(iresmax > 0) {
    hipMemcpy(nres_p, nres, sizeof(INT) * iresmax, hipMemcpyHostToDevice);
    hipMemcpy(mres_p, mres, sizeof(INT) * iresmax, hipMemcpyHostToDevice);

    //hipError_t error;
    //error = hipGetLastError();
    //if(error != hipSuccess) { printf("Before memcopies, cuError: %s\n", hipGetErrorString(error)); }

    qlsum_gpu_iharm_shared<<<blocks, 64>>>(nkx1, nkx2,
					   nky1, nky2,
					   nkdim1, nkdim2,
					   mkdim1,
					   iresmax, nres_p, mres_p,
					   zbeta_p, zbeta_iharm_p,
					   xx_p, yy_p,
					   ealphak_p, ebetak_p, ebk_p,
					   eps_t_p);

    qlsum_gpu_iharm_first<<<blocks, threads>>>(nkx1, nkx2,
					       nky1, nky2,
					       nkdim1, mkdim1,
					       nuper, nupar,
					       iharm, iresmax,
					       lmaxdim,
					       iharm * wcw,
					       sqmut0, dui, eps_t_p,
					       sumb_11_nm_t_p, sumb_31_nm_t_p,
					       nres_p, mres_p,
					       sum2_p, sumkx2_p, sumky2_p,
					       uper_p, upar_p,
					       xkxsav_p, xkysav_p,
					       dfduper_p, npara_sav_p,
					       xkperpn_tmp_p, zetai_p, Jni_p,
					       zetamin_tmp_p, dzetai_tmp_p);

    qlsum_gpu_iharm_second<<<blocks, threads>>>(nkx1, nkx2,
						nky1, nky2,
						nkdim1, mkdim1,
						nuper, nupar,
						iharm, iresmax,
						iharm * wcw,
						sqmut0, dui, eps_t_p,
						sumb_11_nm_t_p, sumb_31_nm_t_p,
						nres_p, mres_p,
						sum_p,
						sumwdot_p, sumwdotkx_p, sumwdotky_p,
						uper_p, upar_p,
						xkxsav_p, xkysav_p,
						dfduper_p, dfdupar_p,
						npara_sav_p,
						xkperpn_tmp_p, zetai_p, Jni_p,
						zetamin_tmp_p, dzetai_tmp_p);
  }

  hipDeviceSynchronize();
}

//Description:
//Provides a wrapper around the GPU reduction
void qlsum_cpu_iharm_finish() {
  dim3 threads(nuper);

  qlsum_gpu_iharm_reduction<<<1, threads>>>(nuper, nupar,
					    sum_wdot_p, sum_fx0_p, sum_fy0_p,
					    b_sum_p, c_sum_p, e_sum_p, f_sum_p,
					    sum_p,
					    sum2_p, sumkx2_p, sumky2_p,
					    sumwdot_p, sumwdotkx_p, sumwdotky_p);

  hipDeviceSynchronize();
}

//Description:
//Provides a wrapper around the GPU zbeta update
void qlsum_cpu_iharm_zbeta_update() {
  dim3 blocks(NUMBLOCKS);

  qlsum_gpu_iharm_zbeta_update<<<blocks, 64>>>((nkx2 - nkx1 + 1) * (nky2 - nky1 + 1), zbeta_p, zbeta_iharm_p);

  hipDeviceSynchronize();
}

//Description:
//Zeroes all the output values and copies over all the CPU generated arrays
void qlsum_cpu_nharm_setup(REAL_H *uper_, REAL_H *upar_,
			   REAL_H *dfduper_, REAL_H *dfdupar_,
			   REAL_H *dui_, REAL_H *wcw_,
			   REAL_H *sqmut0_,
			   REAL_H *npara_sav_, REAL_H *Jni_,
			   REAL_H *xkperpn_tmp_, REAL_H *zetamin_tmp_,
			   REAL_H *zetai_, REAL_H *dzetai_tmp_,
			   CMPLX_H *xx_, CMPLX_H *yy_,
			   CMPLX_H *zbeta, CMPLX_H *zbeta_iharm) {
  //Zero accumulators
  hipMemset(sum_wdot_p, 0, sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMemset(sum_fx0_p, 0, sizeof(CMPLX) * nuper * NUMBLOCKS);
  hipMemset(sum_fy0_p, 0, sizeof(CMPLX) * nuper * NUMBLOCKS);

  hipMemset(b_sum_p, 0, sizeof(CMPLX) * nuper * nupar * NUMBLOCKS);
  hipMemset(c_sum_p, 0, sizeof(CMPLX) * nuper * nupar * NUMBLOCKS);
  hipMemset(e_sum_p, 0, sizeof(CMPLX) * nuper * nupar * NUMBLOCKS);
  hipMemset(f_sum_p, 0, sizeof(CMPLX) * nuper * nupar * NUMBLOCKS);

  //Copy over useful arrays
  cudaMemcpyD2SH2D(xx_p, xx_, 2 * (nkx2 - nkx1 + 1));
  cudaMemcpyD2SH2D(yy_p, yy_, 2 * (nky2 - nky1 + 1));
  cudaMemcpyD2SH2D(uper_p, uper_, nuper);
  cudaMemcpyD2SH2D(upar_p, upar_, nupar);
  cudaMemcpyD2SH2D(zbeta_p, zbeta, 2 * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1));
  cudaMemcpyD2SH2D(zbeta_iharm_p, zbeta_iharm, 2 * (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1));

  cudaMemcpyD2SH2D(xkperpn_tmp_p, xkperpn_tmp_, (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1));
  cudaMemcpyD2SH2D(zetai_p, zetai_, (nzeta + 1));
  cudaMemcpyD2SH2D(Jni_p, Jni_, nuper * (2 * lmaxdim + 1) * (nzeta + 1));
  cudaMemcpyD2SH2D(zetamin_tmp_p, zetamin_tmp_, nuper);
  cudaMemcpyD2SH2D(dzetai_tmp_p, dzetai_tmp_, nuper);

  cudaMemcpyD2SH2D(dfduper_p, dfduper_, nuper * nupar);
  cudaMemcpyD2SH2D(dfdupar_p, dfdupar_, nuper * nupar);
  cudaMemcpyD2SH2D(npara_sav_p, npara_sav_, (nkx2 - nkx1 + 1) * (nky2 - nky1 + 1));

  //Set single values
  sqmut0 = *sqmut0_;
  dui = *dui_;
  wcw = *wcw_;
}

//Description:
//Copies the output values back to the CPU
void qlsum_cpu_nharm_finish(CMPLX_H *sum_wdot_, CMPLX_H *sum_fx0_, CMPLX_H *sum_fy0_,
			    CMPLX_H *b_sum_, CMPLX_H *c_sum_, CMPLX_H *e_sum_, CMPLX_H *f_sum_) {
  cudaMemcpyS2DD2H(sum_wdot_, sum_wdot_p, 2 * nuper);
  cudaMemcpyS2DD2H(sum_fx0_, sum_fx0_p, 2 * nuper);
  cudaMemcpyS2DD2H(sum_fy0_, sum_fy0_p, 2 * nuper);
    
  cudaMemcpyS2DD2H(b_sum_, b_sum_p, 2 * nuper * nupar);
  cudaMemcpyS2DD2H(c_sum_, c_sum_p, 2 * nuper * nupar);
  cudaMemcpyS2DD2H(e_sum_, e_sum_p, 2 * nuper * nupar);
  cudaMemcpyS2DD2H(f_sum_, f_sum_p, 2 * nuper * nupar);
}
